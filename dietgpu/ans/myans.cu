#include "hip/hip_runtime.h"
/**
 * @file myans.cpp
 * @brief A simple program that demonstrates the use of the ANS codec.
 *
 * This program demonstrates basic utilization of rANS codec from dietgpu that reads data from the disk.
 *
 * @author Boyuan Zhang
 * @date July 9, 2024
 *
 * @version 1.0
 * @license MIT
 */

#include <gtest/gtest.h>
#include <cmath>
#include <random>
#include <vector>
#include <fstream>
#include <iostream>
#include <string>
#include <chrono>

#include "dietgpu/ans/GpuANSCodec.h"
#include "dietgpu/utils/StackDeviceMemory.h"

using namespace dietgpu;

std::vector<std::vector<uint8_t>> readFileToVector(const std::string &filePath)
{
    auto out = std::vector<std::vector<uint8_t>>();

    // Create an input file stream in binary mode
    std::ifstream file(filePath, std::ios::binary);

    // Check if the file was opened successfully
    if (!file)
    {
        throw std::runtime_error("Could not open file: " + filePath);
    }

    // Seek to the end of the file to determine the file size
    file.seekg(0, std::ios::end);
    std::streamsize fileSize = file.tellg();
    file.seekg(0, std::ios::beg);

    // Create a vector with the appropriate size
    std::vector<uint8_t> buffer(fileSize);

    // Read the file into the vector
    if (!file.read(reinterpret_cast<char *>(buffer.data()), fileSize))
    {
        throw std::runtime_error("Error reading file: " + filePath);
    }

    out.push_back(buffer);
    
    return out;
}

std::vector<GpuMemoryReservation<uint8_t>> toDevice(
    StackDeviceMemory &res,
    const std::vector<std::vector<uint8_t>> &vs,
    hipStream_t stream)
{
    auto out = std::vector<GpuMemoryReservation<uint8_t>>();

    for (auto &v : vs)
    {
        out.emplace_back(res.copyAlloc(stream, v, AllocType::Permanent));
    }

    return out;
}

std::vector<std::vector<uint8_t>> toHost(
    StackDeviceMemory &res,
    const std::vector<GpuMemoryReservation<uint8_t>> &vs,
    hipStream_t stream)
{
    auto out = std::vector<std::vector<uint8_t>>();

    for (auto &v : vs)
    {
        out.emplace_back(v.copyToHost(stream));
    }

    return out;
}

std::vector<GpuMemoryReservation<uint8_t>> buffersToDevice(
    StackDeviceMemory &res,
    const std::vector<uint32_t> &sizes,
    hipStream_t stream)
{
    auto out = std::vector<GpuMemoryReservation<uint8_t>>();

    for (auto &s : sizes)
    {
        out.emplace_back(res.alloc<uint8_t>(stream, s, AllocType::Permanent));
    }

    return out;
}

void runRansCodec(
    const std::string &filePath)
{
    // run on a different stream to test stream assignment
    auto stream = CudaStream::makeNonBlocking();

    int prec = 10;

    auto res = makeStackMemory();

    auto batch_host = readFileToVector(filePath);
    auto batch_dev = toDevice(res, batch_host, stream);

    std::vector<uint32_t> batchSizes(1);

    batchSizes[0] = batch_host[0].size();

    int numInBatch = batchSizes.size();
    uint32_t maxSize = 0;
    for (auto v : batchSizes)
    {
        maxSize = std::max(maxSize, v);
    }

    auto outBatchStride = getMaxCompressedSize(maxSize);

    auto inPtrs = std::vector<const void *>(batchSizes.size());
    {
        for (int i = 0; i < inPtrs.size(); ++i)
        {
            inPtrs[i] = batch_dev[i].data();
        }
    }

    auto enc_dev = res.alloc<uint8_t>(stream, numInBatch * outBatchStride);

    auto encPtrs = std::vector<void *>(batchSizes.size());
    for (int i = 0; i < inPtrs.size(); ++i)
    {
        encPtrs[i] = (uint8_t *)enc_dev.data() + i * outBatchStride;
    }

    auto outCompressedSize_dev = res.alloc<uint32_t>(stream, numInBatch);

    hipStreamSynchronize(stream);
    auto cStart = std::chrono::high_resolution_clock::now();

    ansEncodeBatchPointer(
        res,
        ANSCodecConfig(prec, true),
        numInBatch,
        inPtrs.data(),
        batchSizes.data(),
        nullptr,
        encPtrs.data(),
        outCompressedSize_dev.data(),
        stream);
    
    hipStreamSynchronize(stream);
    auto cEnd = std::chrono::high_resolution_clock::now();

    auto encSize = outCompressedSize_dev.copyToHost(stream);
    
    // report the compression ratio
    std::cout << "Compression ratio: " << (double)batch_host[0].size() / encSize[0] << std::endl;

    // report the time taken and compression throughput in GB/s
    std::chrono::duration<double, std::milli> cElapsed = cEnd - cStart;
    std::cout << "Time taken: " << cElapsed.count() << " ms" << std::endl;
    std::cout << "Compression throughput: " << (double)batch_host[0].size() / (cElapsed.count() * 1e6) << " GB/s" << std::endl;

    for (auto v : encSize)
    {
        // Reported compressed sizes in bytes should be a multiple of 16 for aligned
        // packing
        EXPECT_EQ(v % 16, 0);
    }

    // Decode data
    auto dec_dev = buffersToDevice(res, batchSizes, stream);

    auto decPtrs = std::vector<void *>(batchSizes.size());
    for (int i = 0; i < inPtrs.size(); ++i)
    {
        decPtrs[i] = dec_dev[i].data();
    }

    auto outSuccess_dev = res.alloc<uint8_t>(stream, numInBatch);
    auto outSize_dev = res.alloc<uint32_t>(stream, numInBatch);

    hipStreamSynchronize(stream);
    auto dStart = std::chrono::high_resolution_clock::now();

    ansDecodeBatchPointer(
        res,
        ANSCodecConfig(prec, true),
        numInBatch,
        (const void **)encPtrs.data(),
        decPtrs.data(),
        batchSizes.data(),
        outSuccess_dev.data(),
        outSize_dev.data(),
        stream);

    hipStreamSynchronize(stream);
    auto dEnd = std::chrono::high_resolution_clock::now();

    // report the time taken in and decompression throughput in GB/s
    std::chrono::duration<double, std::milli> dElapsed = dEnd - dStart;
    std::cout << "Time taken: " << dElapsed.count() << " ms" << std::endl;
    std::cout << "Decompression throughput: " << (double)batch_host[0].size() / (dElapsed.count() * 1e6) << " GB/s" << std::endl;

    auto outSuccess = outSuccess_dev.copyToHost(stream);
    auto outSize = outSize_dev.copyToHost(stream);

    for (int i = 0; i < outSuccess.size(); ++i)
    {
        EXPECT_TRUE(outSuccess[i]);
        EXPECT_EQ(outSize[i], batchSizes[i]);
    }

    auto dec_host = toHost(res, dec_dev, stream);
    EXPECT_EQ(batch_host, dec_host);
}

int main(int argc, char **argv)
{
    // process the argv
    if (argc != 2)
    {
        std::cerr << "Usage: " << argv[0] << " <file_path>" << std::endl;
        return 1;
    }

    // run the codec
    runRansCodec(argv[1]);

    return 0;
}
